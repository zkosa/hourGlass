#include "hip/hip_runtime.h"
#include "particle.h"
#include "boundary_axissymmetric.h"
#include "boundary_planar.h"
#include "hip/hip_runtime.h"

__device__
float static_container::Particle::drag_coefficient_global = 0.5;
__device__
float static_container::Particle::restitution_coeff_global = 0.5;

__device__
float Particle::getRestitutionCoeff() {
	return static_container::Particle::restitution_coeff_global;
}
CUDA_HOSTDEV
float Particle::volume() const {
	return radius * radius * radius * pi * 4.0 / 3.0;
}
CUDA_HOSTDEV
float Particle::mass() const {
	return volume() * density;
}
CUDA_HOSTDEV
float Particle::A() const {
	return radius * radius * pi;
}
CUDA_HOSTDEV
float Particle::CdA() const {
	return 0.5 * A();  // TODO: use getCd()
}
CUDA_HOSTDEV
float Particle::CoR() const {
	return Particle::getRestitutionCoeff();
}

CUDA_HOSTDEV
Vec3d Particle::apply_forces() {
	const Vec3d drag_force = -0.5 * density_medium * CdA() * (vel * abs(vel));
	const Vec3d drag_acc = drag_force / mass();

	return Vec3d(0.0f, -g, 0.0f) + drag_acc; // TODO: readd gravity
}

template<typename Boundary_T>
__device__
void Particle::collideToWall(const Boundary_T *wall) {

	const Vec3d n = wall->getNormal(this);

	Vec3d pos_corr { 0, 0, 0 };
	if (std::abs(n * vel) > SMALL && wall->isPlanar()) { // not parallel, and moving
		// Move outwards along the incoming velocity vector so,
		// that the normal correction component equals to the overlap,
		// This doesn't ensure overlap-less corrected position for curved surfaces,
		// so it is performed only for planar boundaries
		pos_corr = (radius - wall->distanceSigned(this)) / std::abs(n * vel) * vel * (-1);
	} else {
		// If there is no wall normal movement,
		// move in surface normal direction to the touching position
		pos_corr = (radius - wall->distanceSigned(this)) * n;
	}

	// move back to the position when it touched the boundary:
	this->move(pos_corr);

	// correct the velocity to conserve energy (dissipation work is not considered!)
	correctVelocity(pos_corr);

	// revert the wall normal velocity component
	vel = vel - (1 + Particle::getRestitutionCoeff()) * (vel * n) * n;
}

// explicitly instantiating the template instances
template __device__ void Particle::collideToWall<Boundary_axissymmetric>(const Boundary_axissymmetric*);
template __device__ void Particle::collideToWall<Boundary_planar>(const Boundary_planar*);

__device__
void Particle::collideToParticle(Particle *other) {
	Vec3d n = other->pos - this->pos; // distance vector, pointing towards the other particle

	const float distance = abs(n);

	// do not do anything with distant particles:
	if (distance > this->getR() + other->getR()) {
		return;
	}

	n = norm(n); // normalize

	// move back to the positions where they just touched the other:
	const Vec3d pos_corr = -0.5 * n * (this->getR() + other->getR() - distance);
	this->move(pos_corr);
	other->move(-pos_corr);

	correctVelocity(pos_corr);
	other->correctVelocity(-pos_corr);

	exchangeImpulse(other);
}

__host__ __device__
void Particle::correctVelocity(const Vec3d &pos_corr) {
	// correct the velocity to conserve energy (dissipation work is not considered!)
	if (vel * vel + 2 * Vec3d(0.0f, -g, 0.0f) * pos_corr >= 0.0) {
		vel = std::sqrt(vel * vel + 2 * Vec3d(0.0f, -g, 0.0f) * pos_corr) * norm(vel);
	} else {
		vel = -std::sqrt(-(vel * vel + 2 * Vec3d(0.0f, -g, 0.0f) * pos_corr)) * norm(vel);
	}
}

__device__
void Particle::exchangeImpulse(Particle *other) {
	Vec3d n = other->pos - this->pos; // distance vector, pointing towards the other particle
	n = norm(n); // normalize

	const Vec3d vel_old = vel; // store it for the other particle
	vel = vel_old - n * (n * vel_old)
			+ (mass() - other->mass()) / (mass() + other->mass()) * n
					* (vel_old * n)
			+ 2 * other->mass() / (mass() + other->mass()) * n
					* (other->getV() * n);

	other->setV(
			other->getV() - n * (other->getV() * n)
					+ 2 * mass() / (other->getM() + mass()) * n * (vel_old * n)
					+ (other->mass() - mass()) / (other->mass() + mass()) * n
							* (other->getV() * n));
}

CUDA_HOSTDEV
void Particle::advance(float dt) {
	// velocity Verlet integration:
	const Vec3d new_pos = pos + vel * dt + acc * dt * dt * 0.5;
	const Vec3d new_acc = apply_forces();
	const Vec3d new_vel = vel + 0.5 * (acc + new_acc) * dt;

	pos = new_pos;
	vel = new_vel;
	acc = new_acc;
}

__global__
void particles_advance(float dt, Particle *particles, int number_of_particles) {
	// grid-stride loop
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < number_of_particles;
		i += blockDim.x * gridDim.x)
	{
		particles[i].advance(dt);
	}
}
