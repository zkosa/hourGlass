#include "hip/hip_runtime.h"
#include "particle.h"
#include "cell.h"
#include "hip/hip_runtime.h"
#include <stdio.h> // for printing from the device
#include <iostream>
#include <hip/hip_runtime.h>
#include <> // just for proper indexing, nvcc includes it anyhow


__device__
bool Cell::containsCuda(const Particle *p) const {
	float r = p->getR();

	return (p->getX() + r > bounds.x1 && p->getX() - r < bounds.x2)
			&& (p->getY() + r > bounds.y1 && p->getY() - r < bounds.y2)
			&& (p->getZ() + r > bounds.z1 && p->getZ() - r < bounds.z2);
}

__device__
void Cell::addParticleCuda(const Particle *p, int *particle_IDs_in_cell, int *particle_ID_counter) {
	int old_index = atomicAdd(particle_ID_counter, 1);
	particle_IDs_in_cell[old_index] = p->getID();
}

__device__
int sum_array_up_to_index(const int *array, int index) {
	// we assume that the index is not larger than the number of elements in the array!
	int sum = 0;
	for (int i=0; i<index; i++) {
		sum += array[i];
	}
	return sum;
}

__device__
void Cell::addParticleCudaMultiCell(const Particle *p, const int *IN_number_of_particleIDs, int *OUT_particle_IDs_in_cells, int cell_index, int *OUT_particle_ID_counter) {

	int cell_index_start = sum_array_up_to_index(IN_number_of_particleIDs, cell_index);
	int old_index = atomicAdd(OUT_particle_ID_counter + cell_index, 1); // index within cell
	OUT_particle_IDs_in_cells[cell_index_start + old_index] = p->getID();
}
/*
__global__
void get_number_of_particles_in_cell(int number_of_particles, const Particle *p, Cell *c, int *number_of_particle_IDs) {
	// grid-stride loop
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < number_of_particles;
		i += blockDim.x * gridDim.x)
	{
		if (c->containsCuda(p + i)) {
			atomicAdd(number_of_particle_IDs, 1);
			printf("%i\n", number_of_particle_IDs);
		}
	}
}

__global__
void get_particle_IDs_in_cell(int number_of_particles, const Particle *p, Cell *c, int *particle_IDs_in_cell, int *index_counter) {
	// grid-stride loop
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < number_of_particles;
		i += blockDim.x * gridDim.x)
	{
		if (c->containsCuda(p + i)) {
			c->addParticleCuda(p + i, particle_IDs_in_cell, index_counter);
		}
	}
}

__host__
void Cell::populateCuda(const Particle* device_particles_ptr, int N_particles) {

//	Cell* device_cell_ptr;
//	hipMalloc((void **)&device_cell_ptr, sizeof(Cell));
//	hipMemcpy(device_cell_ptr, this, sizeof(Cell), hipMemcpyHostToDevice);



// calculate the number of outputs:
	int *device_number_of_particle_IDs = 0;
	CHECK_CUDA( hipMalloc((void **)&device_number_of_particle_IDs, sizeof(int)) );
	CHECK_CUDA( hipMemset(device_number_of_particle_IDs, 0, sizeof(int)) );


	int threads = 256; // recommended first value, must not be larger than 1024
	int blocks = ceil(float(N_particles)/threads);
	// calling function to be run on the GPU:
	get_number_of_particles_in_cell<<<blocks,threads>>>(
			N_particles,
			device_particles_ptr,
			this, // when called from a device cell?  // device_cell_ptr
			device_number_of_particle_IDs
			);
	hipDeviceSynchronize();

	int host_number_of_particle_IDs = 0;
	CHECK_CUDA( hipMemcpy( &host_number_of_particle_IDs,
				device_number_of_particle_IDs,
				sizeof(int),
				hipMemcpyDeviceToHost
				) );
	//hipDeviceSynchronize();

	//if(host_number_of_particle_IDs > 0)

// allocate memory and get the particles after getting to know the size:
	int *device_index_counter;
	CHECK_CUDA( hipMalloc((void **)&device_index_counter, sizeof(int)) );
	CHECK_CUDA( hipMemset(device_index_counter, 0, sizeof(int)) );

	int *device_particle_IDs_in_cell;
	int max_number_of_particles_in_the_cell = host_number_of_particle_IDs; // use the exact, calculated value
	CHECK_CUDA( hipMalloc((void **)&device_particle_IDs_in_cell, max_number_of_particles_in_the_cell*sizeof(int)) );
	CHECK_CUDA( hipMemset(device_particle_IDs_in_cell, -1, max_number_of_particles_in_the_cell*sizeof(int)) );// zero could be a particle ID, so use something obviously not particle id


	get_particle_IDs_in_cell<<<blocks,threads>>>(
			N_particles,
			device_particles_ptr,
			this, // when called from a device cell?  // device_cell_ptr,
			device_particle_IDs_in_cell,
			device_index_counter
			);
	//hipDeviceSynchronize(); // TODO: try to move it one layer higher (from within cell to within scene level, to reduce number of synchronizations)

// copy the resultant cell IDs into the host Cell::particle_IDs vector
	particle_IDs.resize(host_number_of_particle_IDs);
	CHECK_CUDA( hipMemcpy( particle_IDs.data(),
				device_particle_IDs_in_cell,
				host_number_of_particle_IDs * sizeof(int),
				hipMemcpyDeviceToHost
				) );

	//hipFree(device_cell_ptr);
	CHECK_CUDA( hipFree(device_particle_IDs_in_cell) );
	CHECK_CUDA( hipFree(device_number_of_particle_IDs) );

	//hipDeviceReset();
}
*/
