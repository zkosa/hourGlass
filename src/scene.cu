#include "scene.h"
//#include "mainwindow.h"

void Scene::hostToDevice() {
	// TODO: lock data on host!

	int N_particles = particles.size();
	hipMalloc((void **)&device_particles_ptr, N_particles*sizeof(Particle));
	hipMemcpy(device_particles_ptr, &particles[0], N_particles*sizeof(Particle), hipMemcpyHostToDevice);

	int N_cells = cells.size();
	hipMalloc((void **)&device_cells_ptr, N_cells*sizeof(Cell));
	hipMemcpy(device_cells_ptr, &cells[0], N_cells*sizeof(Cell), hipMemcpyHostToDevice);

	int N_boundaries_ax = boundaries_ax.size();
	hipMalloc((void **)&device_boundaries_ax_ptr, N_boundaries_ax*sizeof(Boundary_axissymmetric));
	hipMemcpy(device_boundaries_ax_ptr, &boundaries_ax[0], N_boundaries_ax*sizeof(Boundary_axissymmetric), hipMemcpyHostToDevice);

	int N_boundaries_pl = boundaries_pl.size();
	hipMalloc((void **)&device_boundaries_pl_ptr, N_boundaries_pl*sizeof(Boundary_planar));
	hipMemcpy(device_boundaries_pl_ptr, &boundaries_pl[0], N_boundaries_pl*sizeof(Boundary_planar), hipMemcpyHostToDevice);

}

void Scene::deviceToHost() {

	// copy the particles back for display purposes
	int N_particles = particles.size();
	hipMemcpy( particles.data(),
				device_particles_ptr,
				N_particles*sizeof(Particle),
				hipMemcpyDeviceToHost
				);

	// cell geometry does not change, particle_IDs are not needed on host --> no need to copy

	// boundaries do not change (can it be enforced???) --> no need to copy


	hipFree(device_particles_ptr);
	hipFree(device_cells_ptr);
	hipFree(device_boundaries_ax_ptr);
	hipFree(device_boundaries_pl_ptr);

	// TODO: unlock data on host
}

void Scene::populateCellsCuda() {

}

void Scene::advanceCuda() {
	if (benchmark_mode && simulation_time >= benchmark_simulation_time) { // in benchmark mode the simulation time is fixed
		/*
		 //#error "You must build your code with position independent code if Qt was built with -reduce-relocations.
		if (viewer != nullptr) {
			viewer->wrapStopButtonClicked();
		} else { // do not call the GUI stuff when we are GUI-less
			setFinished();
		}
		*/

		setFinished(); // fix
		printf("The benchmark has been finished.\n");
	} else {
		simulation_time += time_step;
		for (auto &p : particles) {
			//p.advance(time_step);
		}
	}
	advanceCounter();
}

void Scene::collideWithBoundariesCellsCuda() {

}

void Scene::collideParticlesCellsCuda() {

}
