#include "hip/hip_runtime.h"
#include "scene.h"
#include "mainwindow.h"
#include <> // just for proper indexing, nvcc includes it anyhow


void Scene::hostToDevice() {
	// TODO: lock data on host!

	int N_particles = particles.size();
	CHECK_CUDA( hipMalloc((void **)&device_particles_ptr, N_particles*sizeof(Particle)) );
	CHECK_CUDA( hipMemcpy(device_particles_ptr, &particles[0],
				N_particles*sizeof(Particle),
				hipMemcpyHostToDevice) );

	int N_cells = cells.size();
	CHECK_CUDA( hipMalloc((void **)&device_cells_ptr, N_cells*sizeof(Cell)) );
	CHECK_CUDA( hipMemcpy( device_cells_ptr, &cells[0],
							N_cells*sizeof(Cell),
							hipMemcpyHostToDevice) );

	int N_boundaries_ax = boundaries_ax.size();
	CHECK_CUDA( hipMalloc((void **)&device_boundaries_ax_ptr, N_boundaries_ax*sizeof(Boundary_axissymmetric)) );
	CHECK_CUDA( hipMemcpy( device_boundaries_ax_ptr,
							&boundaries_ax[0],
							N_boundaries_ax*sizeof(Boundary_axissymmetric),
							hipMemcpyHostToDevice) );
	// address of function handle is not valid on the device --> recreate it:
	initializeFunctionHandle<<<1,1>>>(device_boundaries_ax_ptr); CHECK_CUDA_POST

	int N_boundaries_pl = boundaries_pl.size();
	CHECK_CUDA( hipMalloc((void **)&device_boundaries_pl_ptr, N_boundaries_pl*sizeof(Boundary_planar)) );
	CHECK_CUDA( hipMemcpy( device_boundaries_pl_ptr,
							&boundaries_pl[0],
							N_boundaries_pl*sizeof(Boundary_planar),
							hipMemcpyHostToDevice) );

}

void Scene::deviceToHost() {

	// copy the particles back for display purposes
	int N_particles = particles.size();
	//CHECK_CUDA_POINTER( device_particles_ptr );
	CHECK_CUDA( hipMemcpy( particles.data(),
				device_particles_ptr,
				N_particles*sizeof(Particle),
				hipMemcpyDeviceToHost
				) );

	// TODO: protect against overwriting (freeing after copy should do it (?))
	/* it has been copied in Scene::populateCellsCuda() */
//	int N_cells = cells.size();
//	CHECK_CUDA( hipMemcpy( cells.data(),
//				device_cells_ptr,
//				N_cells*sizeof(Cell), // TODO: how does it know the changed amount of particle IDS, stored in a vector? (resize particle_IDS?)
//				hipMemcpyDeviceToHost
//				) );

	// boundaries do not change (can it be enforced???) --> no need to copy


	CHECK_CUDA( hipFree(device_particles_ptr) );
	CHECK_CUDA( hipFree(device_cells_ptr) );
	CHECK_CUDA( hipFree(device_boundaries_ax_ptr) );
	CHECK_CUDA( hipFree(device_boundaries_pl_ptr) );

	// TODO: unlock data on host
}

__global__
void get_number_of_particles_per_cell(
		int number_of_particles, const Particle *p,
		int number_of_cells, const Cell *c,
		int *number_of_particle_IDs_per_cell
		)
{
	// nested (2D) grid-stride loop
	int index_particle = blockIdx.x * blockDim.x + threadIdx.x;
	int stride_particle = blockDim.x * gridDim.x;
	int index_cell = blockIdx.y * blockDim.y + threadIdx.y;
	int stride_cell = blockDim.y * gridDim.y;

	for (int i_c = index_cell;
		i_c < number_of_cells;
		i_c += stride_cell)
	{
		for (int i_p = index_particle;
			i_p < number_of_particles;
			i_p += stride_particle)
		{
			if ((c + i_c)->containsCuda(p + i_p)) {
				atomicAdd(number_of_particle_IDs_per_cell + i_c, 1);
			}
		}
	}
}

__global__
void get_particle_IDs_in_cells(
		int number_of_particles, const Particle *p,
		int number_of_cells, Cell *c,
		const int *IN_number_of_particleIDs, // per cell, as input???
		int *OUT_particle_IDs_in_cells,
		int *OUT_particle_ID_counter // per cell, for counting
		)
{
	// nested (2D) grid-stride loop
	int index_particle = blockIdx.x * blockDim.x + threadIdx.x;
	int stride_particle = blockDim.x * gridDim.x;
	int index_cell = blockIdx.y * blockDim.y + threadIdx.y;
	int stride_cell = blockDim.y * gridDim.y;

	for (int i_c = index_cell;
		i_c < number_of_cells;
		i_c += stride_cell)
	{
		for (int i_p = index_particle;
			i_p < number_of_particles;
			i_p += stride_particle)
		{
			if ((c + i_c)->containsCuda(p + i_p)) {
				(c + i_c)->addParticleCudaMultiCell(p + i_p, IN_number_of_particleIDs, OUT_particle_IDs_in_cells, i_c, OUT_particle_ID_counter);
			}
		}
	}
}

void Scene::populateCellsCuda() {

	// this->clearCells(); // do we need something like this?

	int N_cells = cells.size();
	int N_particles = particles.size();

	dim3 threads(std::min(N_cells, 1024), 1); // all cells are within a block with usual number of cells
	dim3 blocks((N_cells + threads.x - 1)/threads.x, (N_particles + threads.y - 1)/threads.y);
	// std::cout << blocks.x << "x" << blocks.y << " X " << threads.x << "x" << threads.y << std::endl;

	int *device_number_of_particle_IDs_per_cell;
	CHECK_CUDA( hipMalloc((void **)&device_number_of_particle_IDs_per_cell, sizeof(int)*N_cells) );
	CHECK_CUDA( hipMemset(device_number_of_particle_IDs_per_cell, 0, sizeof(int)*N_cells) );

	get_number_of_particles_per_cell<<<blocks,threads>>>(
			N_particles,
			device_particles_ptr,
			N_cells,
			device_cells_ptr,
			device_number_of_particle_IDs_per_cell); CHECK_CUDA_POST


	std::vector<int> host_number_of_particle_IDs_per_cell(N_cells);
	CHECK_CUDA( hipMemcpy( host_number_of_particle_IDs_per_cell.data(),
				device_number_of_particle_IDs_per_cell,
				sizeof(int)*N_cells,
				hipMemcpyDeviceToHost
				) );

	int total_number_of_IDs_in_cells = std::accumulate(
			host_number_of_particle_IDs_per_cell.begin(),
			host_number_of_particle_IDs_per_cell.end(),
			0);

	int *device_particle_IDs_per_cell;
	CHECK_CUDA( hipMalloc((void **)&device_particle_IDs_per_cell, sizeof(int)*total_number_of_IDs_in_cells) );
	CHECK_CUDA( hipMemset(device_particle_IDs_per_cell, 0, sizeof(int)*total_number_of_IDs_in_cells) );

	int *device_indices_counter;
	CHECK_CUDA( hipMalloc((void **)&device_indices_counter, sizeof(int)*N_cells) );
	CHECK_CUDA( hipMemset(device_indices_counter, 0, sizeof(int)*N_cells) );

	get_particle_IDs_in_cells<<<blocks,threads>>>(
			N_particles, device_particles_ptr,
			N_cells, device_cells_ptr,
			device_number_of_particle_IDs_per_cell, // input
			device_particle_IDs_per_cell, // output
			device_indices_counter // output, for debugging
			); CHECK_CUDA_POST

//#define CHECK
#ifdef CHECK
	std::vector<int> host_number_of_particle_IDs_per_cell_second_kernel(N_cells);
	CHECK_CUDA( hipMemcpy( host_number_of_particle_IDs_per_cell_second_kernel.data(),
				device_indices_counter,
				sizeof(int)*N_cells,
				hipMemcpyDeviceToHost
				) );

	int total_number_of_IDs_in_cells_second_kernel = std::accumulate(
			host_number_of_particle_IDs_per_cell_second_kernel.begin(),
			host_number_of_particle_IDs_per_cell_second_kernel.end(),
			0);

	if (total_number_of_IDs_in_cells != total_number_of_IDs_in_cells_second_kernel) {
		std::cout << "number of particle IDs does not match between the two kernels: "
				<< total_number_of_IDs_in_cells << " != "
				<< total_number_of_IDs_in_cells_second_kernel << std::endl;

		std::exit(EXIT_FAILURE); // causes trouble in testing
	} else {
		std::cout << "first and second count gives the same number: " << total_number_of_IDs_in_cells << ": fine" << std::endl;
	}
#endif
#undef CHECK

	// copy the collected particle IDs into the cells in the device
	// it is useful for testing, but the target is to keep everything on the device!
	int array_index = 0;
	for (int cell_ID=0; cell_ID<N_cells; cell_ID++) {
		size_t number_of_elements = host_number_of_particle_IDs_per_cell[cell_ID];
		cells[cell_ID].getParticleIDs().resize(number_of_elements);
		CHECK_CUDA( hipMemcpy( cells[cell_ID].getParticleIDs().data(),
					device_particle_IDs_per_cell + array_index,
					sizeof(int)*number_of_elements,
					hipMemcpyDeviceToHost
					) );
		array_index = array_index + number_of_elements;
	}

}

void Scene::advanceCuda() {
	if (benchmark_mode && simulation_time >= benchmark_simulation_time) { // in benchmark mode the simulation time is fixed
		if (viewer != nullptr) {
			// compilation difficulties when linking Qt to device code!
			// error "You must build your code with position independent code if Qt was built with -reduce-relocations.
			viewer->wrapStopButtonClicked();
		} else { // do not call the GUI stuff when we are GUI-less
			setFinished();
		}
	} else {
		simulation_time += time_step;
		int N_particles = particles.size();
		int threads = 256; // recommended first value, must not be larger than 1024
		int blocks = ceil(float(N_particles)/threads);
		// launching kernel on the GPU:
		particles_advance<<<blocks,threads>>>(time_step, device_particles_ptr, N_particles);
	}
	advanceCounter();
}

__global__
void collide_with_boundaries(
		Particle *p, int number_of_particles,
		const Boundary_axissymmetric *boundaries_ax_ptr, int N_boundaries_ax,
		const Boundary_planar *boundaries_pl_ptr, int N_boundaries_pl
		)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i_p = index; i_p < number_of_particles; i_p += stride ) {
		for (int i_b = 0; i_b<N_boundaries_ax; i_b += 1) {

//			(boundaries_ax_ptr + i_b)->distanceDev(p->cGetPos()); CUDA_HELLO; // fine
//			(boundaries_ax_ptr + i_b)->distanceDev(p); CUDA_HELLO; // fail --> IT DOES NOT WORK with particle!

			if ((boundaries_ax_ptr + i_b)->distanceDev((p + i_p)->cGetPos()) < (p + i_p)->getR()) {
				(p + i_p)->collideToWall(boundaries_ax_ptr + i_b);
				// to_be_collided.emplace_back(p, b); // more sophisticated is used on the CPU!!!
			}
		}
	}
// TODO: implement for planar too

}

void Scene::collideWithBoundariesCellsCuda() {
	// number of collision checks:
	// cells (with boundaries) * boundaries * particles_icell = ~ 100 * 2 * 5000/100 = 50 000
	// here the only benefit from the cells that we have to collide only those particles which are in a cell with boundaries

//	auto particle_IDs = getIDsOfParticlesInCellsWithBoundary();
//
//	int* device_particle_IDs;
//	int n = particle_IDs.size();
//
//	CHECK_CUDA( hipMalloc((void **)&device_particle_IDs, n*sizeof(int)) );
//	CHECK_CUDA( hipMemcpy( device_particle_IDs, particle_IDs.data(),
//							n*sizeof(int),
//							hipMemcpyHostToDevice) );
	int N_particles = particles.size();
	int N_boundaries_ax = boundaries_ax.size();
	int N_boundaries_pl = boundaries_pl.size();

	dim3 threads(std::min(N_particles, 256), 1); // all cells are within a block with usual number of cells
	dim3 blocks((N_particles + threads.x - 1)/threads.x, 1);
	//std::cout << blocks.x << "x" << blocks.y << " X " << threads.x << "x" << threads.y << std::endl;
	collide_with_boundaries<<<blocks, threads>>>(
			device_particles_ptr, N_particles,
			device_boundaries_ax_ptr, N_boundaries_ax,
			device_boundaries_pl_ptr, N_boundaries_pl
			); CHECK_CUDA_POST

	// TODO: boundaries constant

}

void Scene::collideParticlesCellsCuda() {

}
