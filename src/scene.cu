#include "hip/hip_runtime.h"
#include "scene.h"
#include "mainwindow.h"
#include <> // just for proper indexing, nvcc includes it anyhow


void Scene::hostToDevice() {
	// TODO: lock data on host!

	int N_particles = particles.size();
	CHECK_CUDA( hipMalloc((void **)&device_particles_ptr, N_particles*sizeof(Particle)) );
	CHECK_CUDA( hipMemcpy(device_particles_ptr, &particles[0],
				N_particles*sizeof(Particle),
				hipMemcpyHostToDevice) );

	int N_cells = cells.size();
	CHECK_CUDA( hipMalloc((void **)&device_cells_ptr, N_cells*sizeof(Cell)) );
	CHECK_CUDA( hipMemcpy( device_cells_ptr, &cells[0],
							N_cells*sizeof(Cell),
							hipMemcpyHostToDevice) );

	int N_boundaries_ax = boundaries_ax.size();
	CHECK_CUDA( hipMalloc((void **)&device_boundaries_ax_ptr, N_boundaries_ax*sizeof(Boundary_axissymmetric)) );
	CHECK_CUDA( hipMemcpy( device_boundaries_ax_ptr,
							&boundaries_ax[0],
							N_boundaries_ax*sizeof(Boundary_axissymmetric),
							hipMemcpyHostToDevice) );
	// address of function handle is not valid on the device --> recreate it:
	initializeFunctionHandle<<<1,1>>>(device_boundaries_ax_ptr); CHECK_CUDA_POST

	int N_boundaries_pl = boundaries_pl.size();
	CHECK_CUDA( hipMalloc((void **)&device_boundaries_pl_ptr, N_boundaries_pl*sizeof(Boundary_planar)) );
	CHECK_CUDA( hipMemcpy( device_boundaries_pl_ptr,
							&boundaries_pl[0],
							N_boundaries_pl*sizeof(Boundary_planar),
							hipMemcpyHostToDevice) );

}

void Scene::deviceToHost() {

	// copy the particles back for display purposes
	int N_particles = particles.size();
	CHECK_CUDA_POINTER( device_particles_ptr );
	CHECK_CUDA( hipMemcpy( particles.data(),
				device_particles_ptr,
				N_particles*sizeof(Particle),
				hipMemcpyDeviceToHost
				) );

	// TODO: protect against overwriting (freeing after copy should do it (?))
	/* it has been copied in Scene::populateCellsCuda() */
//	int N_cells = cells.size();
//	CHECK_CUDA( hipMemcpy( cells.data(),
//				device_cells_ptr,
//				N_cells*sizeof(Cell), // TODO: how does it know the changed amount of particle IDS, stored in a vector? (resize particle_IDS?)
//				hipMemcpyDeviceToHost
//				) );

	// boundaries do not change (can it be enforced???) --> no need to copy


	CHECK_CUDA( hipFree(device_particles_ptr) );
	CHECK_CUDA( hipFree(device_cells_ptr) );
	CHECK_CUDA( hipFree(device_boundaries_ax_ptr) );
	CHECK_CUDA( hipFree(device_boundaries_pl_ptr) );

	// TODO: unlock data on host
}

__global__
void get_number_of_particles_per_cell(
		int number_of_particles, const Particle *p,
		int number_of_cells, const Cell *c,
		int *number_of_particle_IDs_per_cell
		)
{
	// nested (2D) grid-stride loop
	int index_particle = blockIdx.x * blockDim.x + threadIdx.x;
	int stride_particle = blockDim.x * gridDim.x;
	int index_cell = blockIdx.y * blockDim.y + threadIdx.y;
	int stride_cell = blockDim.y * gridDim.y;

	for (int i_c = index_cell;
		i_c < number_of_cells;
		i_c += stride_cell)
	{
		for (int i_p = index_particle;
			i_p < number_of_particles;
			i_p += stride_particle)
		{
			if ((c + i_c)->containsCuda(p + i_p)) {
				atomicAdd(number_of_particle_IDs_per_cell + i_c, 1);
			}
		}
	}
}

__global__
void get_particle_IDs_in_cells(
		int number_of_particles, const Particle *p,
		int number_of_cells, Cell *c,
		const int *IN_number_of_particleIDs, // per cell, as input???
		int *OUT_particle_IDs_in_cells,
		int *OUT_particle_ID_counter // per cell, for counting
		)
{
	// nested (2D) grid-stride loop
	int index_particle = blockIdx.x * blockDim.x + threadIdx.x;
	int stride_particle = blockDim.x * gridDim.x;
	int index_cell = blockIdx.y * blockDim.y + threadIdx.y;
	int stride_cell = blockDim.y * gridDim.y;

	for (int i_c = index_cell;
		i_c < number_of_cells;
		i_c += stride_cell)
	{
		for (int i_p = index_particle;
			i_p < number_of_particles;
			i_p += stride_particle)
		{
			if ((c + i_c)->containsCuda(p + i_p)) {
				(c + i_c)->addParticleCudaMultiCell(p + i_p, IN_number_of_particleIDs, OUT_particle_IDs_in_cells, i_c, OUT_particle_ID_counter);
			}
		}
	}
}

void Scene::populateCellsCuda() {

	// this->clearCells(); // will we need something like this?

	int N_cells = cells.size();
	int N_particles = particles.size();
//	for(int i=0; i<N_cells; i++) {
//		device_cells_ptr[i].populateCuda(device_particles_ptr, N_particles);
//	}

	dim3 threads(std::min(N_cells, 1024), 1); // all cells are within a block with usual number of cells
	dim3 blocks((N_cells + threads.x - 1)/threads.x, (N_particles + threads.y - 1)/threads.y);
	// std::cout << blocks.x << "x" << blocks.y << " X " << threads.x << "x" << threads.y << std::endl;

	int *device_number_of_particle_IDs_per_cell;
	CHECK_CUDA( hipMalloc((void **)&device_number_of_particle_IDs_per_cell, sizeof(int)*N_cells) );
	CHECK_CUDA( hipMemset(device_number_of_particle_IDs_per_cell, 0, sizeof(int)*N_cells) );
	get_number_of_particles_per_cell<<<blocks,threads>>>(
			N_particles,
			device_particles_ptr,
			N_cells,
			device_cells_ptr,
			device_number_of_particle_IDs_per_cell); CHECK_CUDA_POST

	//hipDeviceSynchronize();

	std::vector<int> host_number_of_particle_IDs_per_cell(N_cells);
	CHECK_CUDA( hipMemcpy( host_number_of_particle_IDs_per_cell.data(),
				device_number_of_particle_IDs_per_cell,
				sizeof(int)*N_cells,
				hipMemcpyDeviceToHost
				) );
//	for (auto const& n : host_number_of_particle_IDs_per_cell) {
//			std::cout << n << '\t';
//	} std::cout << std::endl;

/*
	for (auto& ids: host_number_of_particle_IDs_per_cell) {
		std::cout << ids << '\t';
	}
	std::cout << std::endl;
	for (auto& p: particles) {
		std::cout << p.getX() << ',' << p.getY() <<'\t';
	}
	std::cout << std::endl;
	for (auto& c: cells) {
		std::cout << c.getCenter().x << ','<< c.getCenter().y <<'\t';
	}
	std::cout << std::endl;
*/
	int total_number_of_IDs_in_cells = std::accumulate(
			host_number_of_particle_IDs_per_cell.begin(),
			host_number_of_particle_IDs_per_cell.end(),
			0);

	int *device_particle_IDs_per_cell;
	CHECK_CUDA( hipMalloc((void **)&device_particle_IDs_per_cell, sizeof(int)*total_number_of_IDs_in_cells) );
	CHECK_CUDA( hipMemset(device_particle_IDs_per_cell, 0, sizeof(int)*total_number_of_IDs_in_cells) );

	int *device_indices_counter;
	CHECK_CUDA( hipMalloc((void **)&device_indices_counter, sizeof(int)*N_cells) );
	CHECK_CUDA( hipMemset(device_indices_counter, 0, sizeof(int)*N_cells) );

	get_particle_IDs_in_cells<<<blocks,threads>>>(
			N_particles, device_particles_ptr,
			N_cells, device_cells_ptr,
			device_number_of_particle_IDs_per_cell, // input
			device_particle_IDs_per_cell, // output
			device_indices_counter // output, for debugging
			); CHECK_CUDA_POST

/*
#define CHECK
#ifdef CHECK
	std::vector<int> host_number_of_particle_IDs_per_cell_second_kernel(N_cells);
	CHECK_CUDA( hipMemcpy( host_number_of_particle_IDs_per_cell_second_kernel.data(),
				device_indices_counter,
				sizeof(int)*N_cells,
				hipMemcpyDeviceToHost
				) );

	int total_number_of_IDs_in_cells_second_kernel = std::accumulate(
			host_number_of_particle_IDs_per_cell_second_kernel.begin(),
			host_number_of_particle_IDs_per_cell_second_kernel.end(),
			0);

	if (total_number_of_IDs_in_cells != total_number_of_IDs_in_cells_second_kernel) {
		std::cout << "number of particle IDs does not match between the two kernels: "
				<< total_number_of_IDs_in_cells << " != "
				<< total_number_of_IDs_in_cells_second_kernel << std::endl;

		std::exit(EXIT_FAILURE); // causes trouble in testing
	} else {
		std::cout << "first and second count gives the same number: " << total_number_of_IDs_in_cells << ": fine" << std::endl;
	}
#endif
#undef CHECK
*/
/*
	// TODO: transfer the results to the right place...
	// store the results here until no better solution has been implemented
	//worst case: copy back to the host Cell objects (DeviceToHost copy, not preferred, but currently needed)
	// ideally device to device copy
	std::vector<std::vector<int>> cell_particle_IDs(N_cells);
	int array_index = 0;
	int cell_ID = 0;
	for (auto const & number_of_particles_in_cell : host_number_of_particle_IDs_per_cell) {
		int chunk = number_of_particles_in_cell;
		cell_particle_IDs[cell_ID].resize(chunk);
		CHECK_CUDA( hipMemcpy( cell_particle_IDs[cell_ID].data(),
					device_particle_IDs_per_cell + array_index,
					sizeof(int)*chunk,
					hipMemcpyDeviceToHost
					) );
		array_index = array_index + chunk;
		cell_ID = cell_ID + 1;
	}

	std::cout<< "----" << std::endl;
	for (auto const& IDs: cell_particle_IDs) {
		for (auto const& ID : IDs) {
				std::cout << ID << '\t';
		}
		std::cout << std::endl;
	}
	std::cout<< "----" << std::endl;
*/
	// copy the collected particle IDs into the cells in the device
	// it is useful for testing, but the target is to keep everything on the device!
	int array_index = 0;
	for (int cell_ID=0; cell_ID<N_cells; cell_ID++) {
		size_t number_of_elements = host_number_of_particle_IDs_per_cell[cell_ID];
		cells[cell_ID].getParticleIDs().resize(number_of_elements);
		CHECK_CUDA( hipMemcpy( cells[cell_ID].getParticleIDs().data(),
					device_particle_IDs_per_cell + array_index,
					sizeof(int)*number_of_elements,
					hipMemcpyDeviceToHost
					) );
		array_index = array_index + number_of_elements;
	}

}

void Scene::advanceCuda() {
	if (benchmark_mode && simulation_time >= benchmark_simulation_time) { // in benchmark mode the simulation time is fixed
		if (viewer != nullptr) {
			// compilation difficulties when linking Qt to device code!
			// error "You must build your code with position independent code if Qt was built with -reduce-relocations.
			viewer->wrapStopButtonClicked();
		} else { // do not call the GUI stuff when we are GUI-less
			setFinished();
		}
	} else {
		simulation_time += time_step;
		int N_particles = particles.size();
		int threads = 256; // recommended first value, must not be larger than 1024
		int blocks = ceil(float(N_particles)/threads);
		// launching kernel on the GPU:
		particles_advance<<<blocks,threads>>>(time_step, device_particles_ptr, N_particles);
	}
	advanceCounter();
}

__global__
void collide_with_boundaries(
		Particle *p, int number_of_particles,
		const Boundary_axissymmetric *boundaries_ax_ptr, int N_boundaries_ax,
		const Boundary_planar *boundaries_pl_ptr, int N_boundaries_pl
		)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i_p = index; i_p < number_of_particles; i_p += stride ) {
		for (int i_b = 0; i_b<N_boundaries_ax; i_b += 1) {

			printf("p: %p\n", (void*)p);
			printf("p->getX(): %f\n", p->getX());
//			boundaries_ax_ptr[i_b];
//			boundaries_ax_ptr[i_b].distanceDev(p); // error already here: out of bounds
//			boundaries_ax_ptr[i_b].distanceDev(p + 1);
//			boundaries_ax_ptr[i_b].distanceDev(p + i_p);
			(boundaries_ax_ptr + i_b);
			p->advance(0.0001f);
			//(boundaries_ax_ptr + i_b)->distanceDev(p); // error already here: out of bounds
			// it must be distanceDev
			//(boundaries_ax_ptr + i_b)->distanceDev( (p + 1)->cGetPos()); // it causes undefined reference problems
			(boundaries_ax_ptr + i_b)->distanceDev(p + 1);
			(boundaries_ax_ptr + i_b)->distanceDev(p + i_p);
			//p[i_p].getR();
//			(boundaries_ax_ptr + i_b)->distanceDev(p->cGetPos()); CUDA_HELLO; // fine
//			(boundaries_ax_ptr + i_b)->distanceDev(p); CUDA_HELLO; // fail --> IT DOES NOT WORK with particle!

			if ((boundaries_ax_ptr + i_b)->distanceDev((p + i_p)->cGetPos()) < (p + i_p)->getR()) {
				(p + i_p)->collideToWall(boundaries_ax_ptr + i_b);
				// to_be_collided.emplace_back(p, b); // more sophisticated is used on the CPU!!!
			}
		}
	}
// TODO: implement for planar too

}

void Scene::collideWithBoundariesCellsCuda() {
	// number of collision checks:
	// cells (with boundaries) * boundaries * particles_icell = ~ 100 * 2 * 5000/100 = 50 000
	// here the only benefit from the cells that we have to collide only those particles which are in a cell with boundaries

//	auto particle_IDs = getIDsOfParticlesInCellsWithBoundary();
//
//	int* device_particle_IDs;
//	int n = particle_IDs.size();
//
//	CHECK_CUDA( hipMalloc((void **)&device_particle_IDs, n*sizeof(int)) );
//	CHECK_CUDA( hipMemcpy( device_particle_IDs, particle_IDs.data(),
//							n*sizeof(int),
//							hipMemcpyHostToDevice) );
	int N_particles = particles.size();
	int N_boundaries_ax = boundaries_ax.size();
	int N_boundaries_pl = boundaries_pl.size();


	CHECK_CUDA_POINTER( device_particles_ptr );
	CHECK_CUDA_POINTER( device_boundaries_ax_ptr );
	CHECK_CUDA_POINTER( device_boundaries_pl_ptr );
	collide_with_boundaries<<<1, N_particles>>>( // TODO: fix
	//collide_with_boundaries<<<1, 1>>>( // DEBUG
			device_particles_ptr, N_particles,
			device_boundaries_ax_ptr, N_boundaries_ax,
			device_boundaries_pl_ptr, N_boundaries_pl
			); CHECK_CUDA_POST

	// TODO: boundaries constant

}

void Scene::collideParticlesCellsCuda() {

}
